#include "hip/hip_runtime.h"
// NV12ToRGBA
// cuda
#include "hip/hip_runtime.h"   
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h" // ��ͷ�ļ����� __syncthreads ()����
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <math.h>

__global__ void YCrCb2RGBConver(unsigned char *pYdata, unsigned char *pUVdata, int stepY, int stepUV, 
	unsigned char *pImgData, int width, int height, int channels)
{
	const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	const int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if (tidx < width && tidy < height)
	{
		int indexY, indexU, indexV;
		unsigned char Y, U, V;
		indexY = tidy * stepY + tidx;
		Y = pYdata[indexY];
		
		if (tidx % 2 == 0)
		{
			indexU = tidy / 2 * stepUV + tidx;
			indexV = tidy / 2 * stepUV + tidx + 1;
			U = pUVdata[indexU];
			V = pUVdata[indexV];
		}
		else if (tidx % 2 == 1)
		{
			indexV = tidy / 2 * stepUV + tidx;
			indexU = tidy / 2 * stepUV + tidx - 1;
			U = pUVdata[indexU];
			V = pUVdata[indexV];
		}

		int index = (tidy*width + tidx) * channels;
		pImgData[index + 0] = unsigned char(Y + 1.402 * (V - 128));
		pImgData[index + 1] = unsigned char(Y - 0.34413 * (U - 128) - 0.71414*(V - 128));
		pImgData[index + 2] = unsigned char(Y + 1.772*(U - 128));
		pImgData[index + 3] = unsigned char(255);
	}
}

extern "C" void NV12TORGBA(unsigned char *pYdata, unsigned char *pUVdata, int stepY, int stepUV,
	unsigned char *pImgData, int width, int height, int channels) {

	int uint = 32;
	dim3 block(uint, uint);
	int wblock = (width + block.x - 1) / block.x;
	int hblock = (height + block.y - 1) / block.y;
	//std::cout << wblock << std::ends << hblock << std::endl;
	dim3 grid(wblock, hblock);

	YCrCb2RGBConver << <grid, block >> >(pYdata, pUVdata, stepY, stepUV, pImgData, width, height, 4);
}




